#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void HelloFromGPU(void) {
	printf("hello from GPU\n");
	return;	
}

void HelloFromCPU(void) {
	printf("hello from CPU\n");
	return;	
}

int main (int argc, const char** argv, const char** env) {
	HelloFromCPU();
	HelloFromGPU<<<2, 5>>>();
	HelloFromCPU();
	return 0;
}
